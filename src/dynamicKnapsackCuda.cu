#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <algorithm>
#include <climits>
#include <iostream>
#include <nlohmann/json.hpp>
#include <vector>

__global__ void dynamicKnapsackItemKernel(int maxW, int maxS,
                                          const int *d_dp_prev, int *d_dp_curr,
                                          int item_weight, int item_size,
                                          int item_value)
{
  int w = blockIdx.x * blockDim.x + threadIdx.x;
  int s = blockIdx.y * blockDim.y + threadIdx.y;

  if (w > maxW || s > maxS)
  {
    return;
  }

  int prev_idx = w * (maxS + 1) + s;
  d_dp_curr[prev_idx] = d_dp_prev[prev_idx];

  if (w >= item_weight && s >= item_size)
  {
    int prev_take_idx = (w - item_weight) * (maxS + 1) + (s - item_size);
    d_dp_curr[prev_idx] =
        max(d_dp_curr[prev_idx], d_dp_prev[prev_take_idx] + item_value);
  }
}

extern "C" int runDynamicKnapsackCuda(int n, int maxW, int maxS,
                                      const int *h_weights, const int *h_sizes,
                                      const int *h_values)
{
  hipEvent_t startEvent, stopEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);
  hipEventRecord(startEvent);

  size_t dp_table_size_bytes = (size_t)(maxW + 1) * (maxS + 1) * sizeof(int);

  int *d_dp_prev;
  int *d_dp_curr;
  hipMalloc(&d_dp_prev, dp_table_size_bytes);
  hipMalloc(&d_dp_curr, dp_table_size_bytes);

  hipMemset(d_dp_prev, 0, dp_table_size_bytes);

  int *d_weights;
  int *d_sizes;
  int *d_values;
  hipMalloc(&d_weights, n * sizeof(int));
  hipMalloc(&d_sizes, n * sizeof(int));
  hipMalloc(&d_values, n * sizeof(int));
  hipMemcpy(d_weights, h_weights, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sizes, h_sizes, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_values, h_values, n * sizeof(int), hipMemcpyHostToDevice);

  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((maxW + 1 + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (maxS + 1 + threadsPerBlock.y - 1) / threadsPerBlock.y);

  for (int i = 0; i < n; ++i)
  {
    dynamicKnapsackItemKernel<<<numBlocks, threadsPerBlock>>>(
        maxW, maxS, d_dp_prev, d_dp_curr, h_weights[i], h_sizes[i],
        h_values[i]);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
      std::cerr << "CUDA error after kernel launch for item " << i << ": "
                << hipGetErrorString(err) << std::endl;

      hipFree(d_dp_prev);
      hipFree(d_dp_curr);
      hipFree(d_weights);
      hipFree(d_sizes);
      hipFree(d_values);
      hipEventDestroy(startEvent);
      hipEventDestroy(stopEvent);
      return -1;
    }
    hipDeviceSynchronize();

    int *temp = d_dp_prev;
    d_dp_prev = d_dp_curr;
    d_dp_curr = temp;
  }

  int result = 0;

  hipMemcpy(&result, d_dp_prev + maxW * (maxS + 1) + maxS, sizeof(int),
             hipMemcpyDeviceToHost);

  hipEventRecord(stopEvent);
  hipEventSynchronize(stopEvent);
  float elapsedMs = 0.0f;
  hipEventElapsedTime(&elapsedMs, startEvent, stopEvent);

  hipFree(d_dp_prev);
  hipFree(d_dp_curr);
  hipFree(d_weights);
  hipFree(d_sizes);
  hipFree(d_values);
  hipEventDestroy(startEvent);
  hipEventDestroy(stopEvent);

  std::cout << "{\"value\": " << result << "}" << std::endl;
  std::cerr << "CUDA Time: " << elapsedMs << " ms" << std::endl;
  return result;
}

int main()
{
  nlohmann::json data;
  std::cin >> data;

  int n = data["n"];
  int maxW = data["maxweight"];
  int maxS = data["maxsize"];
  std::vector<int> weights_vec = data["weights"].get<std::vector<int>>();
  std::vector<int> sizes_vec = data["sizes"].get<std::vector<int>>();
  std::vector<int> values_vec = data["values"].get<std::vector<int>>();

  runDynamicKnapsackCuda(n, maxW, maxS, weights_vec.data(), sizes_vec.data(),
                         values_vec.data());

  return 0;
}
