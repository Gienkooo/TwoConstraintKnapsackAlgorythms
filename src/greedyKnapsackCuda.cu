#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

#include <algorithm>
#include <climits>
#include <iostream>
#include <nlohmann/json.hpp>
#include <vector>

struct ItemCuda {
  int id;
  int weight;
  int size;
  int value;
  double ratio;
};

__global__ void greedyRatioKernel(int n, const int* weights, const int* sizes,
                                  const int* values, double* ratios) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    double denom = (double)weights[idx] + sizes[idx];
    ratios[idx] = denom > 0.0 ? (double)values[idx] / denom : 0.0;
  }
}

extern "C" int runGreedyKnapsackCuda(int n, int maxW, int maxS,
                                     const int* weights, const int* sizes,
                                     const int* values) {
  thrust::host_vector<ItemCuda> h_items(n);
  for (int i = 0; i < n; ++i) {
    h_items[i].id = i;
    h_items[i].weight = weights[i];
    h_items[i].size = sizes[i];
    h_items[i].value = values[i];
    h_items[i].ratio = 0.0;
  }
  int* d_weights;
  int* d_sizes;
  int* d_values;
  double* d_ratios;
  hipMalloc(&d_weights, n * sizeof(int));
  hipMalloc(&d_sizes, n * sizeof(int));
  hipMalloc(&d_values, n * sizeof(int));
  hipMalloc(&d_ratios, n * sizeof(double));
  hipMemcpy(d_weights, weights, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_sizes, sizes, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_values, values, n * sizeof(int), hipMemcpyHostToDevice);
  int blockSize = 256;
  int gridSize = (n + blockSize - 1) / blockSize;
  hipEvent_t startEvent, stopEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);
  hipEventRecord(startEvent);
  greedyRatioKernel<<<gridSize, blockSize>>>(n, d_weights, d_sizes, d_values,
                                             d_ratios);
  hipMemcpy(&(h_items[0].ratio), d_ratios, n * sizeof(double),
             hipMemcpyDeviceToHost);
  hipEventRecord(stopEvent);
  hipEventSynchronize(stopEvent);
  float elapsedMs = 0.0f;
  hipEventElapsedTime(&elapsedMs, startEvent, stopEvent);
  hipFree(d_weights);
  hipFree(d_sizes);
  hipFree(d_values);
  hipFree(d_ratios);
  std::sort(
      h_items.begin(), h_items.end(),
      [](const ItemCuda& a, const ItemCuda& b) { return a.ratio > b.ratio; });
  int currentW = 0, currentS = 0, totalValue = 0;
  for (int i = 0; i < n; ++i) {
    if (currentW + h_items[i].weight <= maxW &&
        currentS + h_items[i].size <= maxS) {
      currentW += h_items[i].weight;
      currentS += h_items[i].size;
      totalValue += h_items[i].value;
    }
  }
  std::cout << totalValue << std::endl;
  std::cerr << "Time: " << elapsedMs << " ms" << std::endl;
  return totalValue;
}

int greedyKnapsackCuda(int n, int maxW, int maxS,
                       const std::vector<int>& weights,
                       const std::vector<int>& sizes,
                       const std::vector<int>& values, float& elapsedMs) {
  hipEvent_t startEvent, stopEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);
  hipEventRecord(startEvent);

  thrust::host_vector<ItemCuda> h_items(n);
  for (int i = 0; i < n; ++i) {
    h_items[i].id = i;
    h_items[i].weight = weights[i];
    h_items[i].size = sizes[i];
    h_items[i].value = values[i];
    double weight_plus_size = static_cast<double>(weights[i]) + sizes[i];
    h_items[i].ratio = (weight_plus_size > 0)
                           ? static_cast<double>(values[i]) / weight_plus_size
                           : -1.0;
  }

  thrust::device_vector<ItemCuda> d_items = h_items;
  thrust::sort(
      d_items.begin(), d_items.end(),
      [](const ItemCuda& a, const ItemCuda& b) { return a.ratio > b.ratio; });

  thrust::host_vector<ItemCuda> sorted_h_items = d_items;

  int currentW = 0;
  int currentS = 0;
  int totalValue = 0;

  for (int i = 0; i < n; ++i) {
    if (currentW + sorted_h_items[i].weight <= maxW &&
        currentS + sorted_h_items[i].size <= maxS) {
      currentW += sorted_h_items[i].weight;
      currentS += sorted_h_items[i].size;
      totalValue += sorted_h_items[i].value;
    }
  }

  hipEventRecord(stopEvent);
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&elapsedMs, startEvent, stopEvent);

  hipEventDestroy(startEvent);
  hipEventDestroy(stopEvent);

  return totalValue;
}

__global__ void gather_fitness_kernel(int pop_size, const int* old_fitness,
                                      const int* sorted_indices,
                                      int* new_fitness) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < pop_size) {
    new_fitness[idx] = old_fitness[sorted_indices[idx]];
  }
}

int main() {
  nlohmann::json data;
  std::cin >> data;
  int n = data["n"];
  int maxW = data["maxweight"];
  int maxS = data["maxsize"];
  std::vector<int> weights_vec = data["weights"].get<std::vector<int>>();
  std::vector<int> sizes_vec = data["sizes"].get<std::vector<int>>();
  std::vector<int> values_vec = data["values"].get<std::vector<int>>();
  float elapsedMs = 0;

  int result = greedyKnapsackCuda(n, maxW, maxS, weights_vec, sizes_vec,
                                  values_vec, elapsedMs);

  std::cout << "{\\" value\\": " << result << "}" << std::endl;
  std::cerr << "CUDA Time: " << elapsedMs << " ms" << std::endl;

  return 0;
}

extern "C" int runGreedyKnapsackCuda(int n, int maxW, int maxS,
                                     const int* h_weights, const int* h_sizes,
                                     const int* h_values) {
  std::vector<int> weights_vec(h_weights, h_weights + n);
  std::vector<int> sizes_vec(h_sizes, h_sizes + n);
  std::vector<int> values_vec(h_values, h_values + n);
  float elapsedMs = 0;
  int result = greedyKnapsackCuda(n, maxW, maxS, weights_vec, sizes_vec,
                                  values_vec, elapsedMs);
  std::cout << "{\\" value\\": " << result << "}" << std::endl;
  std::cerr << "CUDA Time: " << elapsedMs << " ms" << std::endl;
  return result;
}
